#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "main.hh"
#include "solve.hh"
using namespace std;

#define matrix(i, j) arr[(i) *n + (j)]

__global__ void division_kernel(float arr[], int n, int k) {
  auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (k < tid && tid < n) matrix(k, tid) /= matrix(k, k);
}

__global__ void eliminate_kernel(float arr[], int n, int k) {
  auto tx = blockDim.x * blockIdx.x + threadIdx.x;
  if (tx == 0) matrix(k, k) = 1.0;
  for (auto row = k + 1 + blockIdx.x; row < n; row += gridDim.x) {
    auto tid = threadIdx.x;
    for (auto col = k + 1 + tid; col < n; col += blockDim.x)
      matrix(row, col) -= matrix(row, k) * matrix(k, col);
    __syncthreads();
    if (tid == 0) matrix(row, k) = 0;
  }
}

void func(int& ans, float arr[], int n) {
  float* gpu_arr;
  auto   siz = sizeof(float) * n * n;
  if (hipMalloc(&gpu_arr, siz) != hipSuccess)
    cerr << "hipMalloc failed" << endl;
  if (hipMemcpy(gpu_arr, arr, siz, hipMemcpyHostToDevice) != hipSuccess)
    cerr << "hipMemcpyHostToDevice failed" << endl;

  for (int k = 0; k < n; ++k) {
    auto rest = n - k - 1;
    dim3 grid(std::ceil(rest / 1024.0));
    dim3 block(1024);
    division_kernel<<<grid, block>>>(gpu_arr, n, k);
    hipDeviceSynchronize();
    if (auto ret = hipGetLastError(); ret != hipSuccess)
      cerr << "division kernel failed: " << hipGetErrorString(ret) << endl;
    dim3 eliminate_grid(32);
    eliminate_kernel<<<eliminate_grid, block>>>(gpu_arr, n, k);
    hipDeviceSynchronize();
    if (auto ret = hipGetLastError(); ret != hipSuccess)
      cerr << "eliminate kernel failed: " << hipGetErrorString(ret) << endl;
  }
  if (hipMemcpy(arr, gpu_arr, siz, hipMemcpyDeviceToHost) != hipSuccess)
    cerr << "hipMemcpyDeviceToHost failed" << endl;
  hipFree(gpu_arr);
}

#undef matrix

signed main(int argc, char* argv[]) {
  return _main(argc, argv);
}