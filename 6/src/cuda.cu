#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "main.hh"
#include "solve.hh"
using namespace std;

#define matrix(i, j) arr[(i) *n + (j)]

__global__ void division_kernel(float arr[], int n, int k) {
  auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (k < tid && tid < n) matrix(k, tid) /= matrix(k, k);
}

__global__ void eliminate_kernel(float arr[], int n, int k) {
  auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) matrix(k, k) = 1.0;
}

void func(int& ans, float arr[], int n) {
  float* gpu_arr;
  auto   siz = sizeof(float) * n * n;
  if (hipMalloc(&gpu_arr, siz) != hipSuccess)
    cerr << "hipMalloc failed" << endl;
  if (hipMemcpy(gpu_arr, arr, siz, hipMemcpyHostToDevice) != hipSuccess)
    cerr << "hipMemcpyHostToDevice failed" << endl;

  for (int k = 0; k < n; ++k) {
    auto rest = n - k;
    dim3 grid(std::ceil(rest / 1024.0));
    dim3 block(1024);
    division_kernel<<<grid, block>>>(gpu_arr, n, k);
    hipDeviceSynchronize();
    if (auto ret = hipGetLastError(); ret != hipSuccess)
      cerr << "division kernel failed: " << hipGetErrorString(ret) << endl;
    matrix(k, k) = 1.0;
    eliminate_kernel<<<grid, block>>>(gpu_arr, n, k);
    hipDeviceSynchronize();
    if (auto ret = hipGetLastError(); ret != hipSuccess)
      cerr << "eliminate kernel failed: " << hipGetErrorString(ret) << endl;
  }
  if (hipMemcpy(arr, gpu_arr, siz, hipMemcpyDeviceToHost) != hipSuccess)
    cerr << "hipMemcpyDeviceToHost failed" << endl;
  hipFree(gpu_arr);
}

#undef matrix

signed main(int argc, char* argv[]) {
  return _main(argc, argv);
}